#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include <time.h>

#include "utils.h"
#include "hash.h"

#define INPUT_SIZE 52

__global__ void find_nonce_kernel(uint8_t *, uint64_t *target,
                                  int *, uint8_t *, uint64_t *);

int main() {

  freopen("./test.in", "r", stdin);

  char prev_digest_hex_str[65];
  uint64_t target;
  read_inputs(prev_digest_hex_str, &target);

  uint8_t input[INPUT_SIZE];
  time_t timestamp = time(NULL);
  generate_partial_hash_input(input, timestamp, prev_digest_hex_str);

  uint8_t *d_input;
  hipMalloc((void **) &d_input, sizeof(input));
  hipMemcpy(d_input, input, sizeof(input), hipMemcpyHostToDevice);

  uint64_t *d_target;
  hipMalloc((void **) &d_target, sizeof(uint64_t));
  hipMemcpy(d_target, &target, sizeof(target), hipMemcpyHostToDevice);

  int *d_found;
  int found = 0;
  hipMalloc((void **) &d_found, sizeof(int));
  hipMemcpy(d_found, &found, sizeof(int), hipMemcpyHostToDevice);

  uint8_t hash[32];
  uint8_t *d_hash;
  hipMalloc((void **) &d_hash, sizeof(hash));

  uint64_t *d_nonce;
  uint64_t nonce[1] = {0};
  hipMalloc((void **) &d_nonce, sizeof(nonce));

  find_nonce_kernel<<<2, 64>>>(d_input, d_target, d_found, d_hash, d_nonce);

  // Copy input back
  hipMemcpy(input, d_input, sizeof(input), hipMemcpyDeviceToHost);

  // Copy nonce back
  hipMemcpy(nonce, d_nonce, sizeof(nonce), hipMemcpyDeviceToHost);

  // Copy digest/hash back
  hipMemcpy(hash, d_hash, sizeof(hash), hipMemcpyDeviceToHost);

  // Print final output
  printf("Target: %llu\n", target);
  print_final_output(timestamp, nonce[0], hash);

  hipFree(d_input); hipFree(d_target); hipFree(d_found);
  hipFree(d_nonce); hipFree(d_hash);

  return 0;
}

__device__ void print_nonce(uint64_t nonce) {
  printf("%d\n", nonce);
}

__global__ void find_nonce_kernel(uint8_t *g_input, uint64_t *g_target,
                                  int *found, uint8_t *hash, uint64_t *g_nonce) {
  // Copy from global to local
  uint8_t l_input[52];
  uint8_t l_hash[32];

  uint64_t l_target = *g_target;

  for (int i=0; i<NONCE_IDX; i++) l_input[i] = g_input[i];

  size_t total_num_threads = gridDim.x * gridDim.y * gridDim.z
                             * blockDim.x * blockDim.y * blockDim.z;

  size_t block_index_in_grid = blockIdx.x * (gridDim.y * gridDim.z)
                               + blockIdx.y * (gridDim.z) + blockIdx.z;

  size_t thread_index_in_block = threadIdx.x * (blockDim.y * blockDim.z)
                                 + threadIdx.y * (blockDim.z) + threadIdx.z;

  size_t thread_id = block_index_in_grid * (blockDim.x * blockDim.y * blockDim.z)
                     + thread_index_in_block;

  uint64_t nonce = thread_id;

  // Start finding nonce
  while (!*found) {
//    print_nonce(nonce);
    fill_input_with_nonce(l_input, nonce);
    sha256(l_hash, l_input, 52);

    if (check_if_valid_nonce(l_hash, l_target) && *found == 0) {
      int old = atomicAdd(found, 1);
      if (old == 0) {
        // Only one thread can ever do this
        // Copy back input to global memory
        for (int i=0; i<INPUT_SIZE; i++) g_input[i] = l_input[i];

        // Copy hash to global memory
        for (int i=0; i<32; i++) hash[i] = l_hash[i];

        // Copy over nonce
        *g_nonce = nonce;
      }
    }
    nonce += total_num_threads;
  }

}