/*
 * Hello World in CUDA
 *
 * CS3210
 *
 * This program start from "hello world" string and should print "HELLO WORLD"
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N       32 

// #define      DISCRETE

__global__ void hello(char *a, int len)
{
        //int tid = threadIdx.x;
	int block_index_in_grid = blockIdx.x * (gridDim.y * gridDim.z) + blockIdx.y * (gridDim.z) + blockIdx.z;
	int thread_index_in_block = threadIdx.x * (blockDim.y * blockDim.z) + threadIdx.y * (blockDim.z) + threadIdx.z;
	int tid = block_index_in_grid * (blockDim.x * blockDim.y * blockDim.z) + thread_index_in_block;
        if (tid >= len)
                return;
        a[tid] += 'A' - 'a';
}

int main()
{
        // original string
        char a[N] = "hello@world";
        // length
        int len = strlen(a);
        // pointer to the string on device
        char* ad;
        // pointer to the final string on host
        char* ah;
        // CUDA returned error code
        hipError_t rc;


        //allocate space for the string on device (GPU) memory
        hipMalloc((void**)&ad, N);
        hipMemcpy(ad, a, N, hipMemcpyHostToDevice);

        // launch the kernel
	dim3 gridDim(2, 2, 2);
	dim3 blockDim(2, 4);
        hello<<<gridDim, blockDim>>>(ad, len);
        hipDeviceSynchronize();

	// for discrete GPUs, get the data from device memory to host memory
        hipMemcpy(a, ad, N, hipMemcpyDeviceToHost);
        ah = a;

        // was there any error?
        rc = hipGetLastError();
        if (rc != hipSuccess)
                printf("Last CUDA error %s\n", hipGetErrorString(rc));

        // print final string
        printf("%s!\n", ah);

        // free memory
        hipFree(ad);

        return 0;
}

